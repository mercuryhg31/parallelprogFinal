#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

extern "C" 
{
    int mandelbrot(Complex c, unsigned int iterations, int blockNumb, int threadsNumb);
}


__global__ typedef struct {
    double real;
    double imag;
} Complex;


int mandelbrot(Complex *c, unsigned int iterations, int threadsNumb, ){
    size_t array_size = sizeof(c);
    mandelbrotKernel<<<array_size/threadsNumb,threadsNumb>>>(c, iterations); //blocks = arraySize/threads and the threadcount is specified by user
    hipDeviceSynchronize(); //sync final result 
}


__global__ int mandelbrotKernel(Complex c, unsigned int iterations) {
    Complex z = {0, 0};

    for (int i = 0; i < iterations; i++) {
        double z_real_sq = z.real * z.real;
        double z_imag_sq = z.imag * z.imag;

        if (z_real_sq + z_imag_sq > 4.0) {
            return i; // escaped
        }

        double z_real_temp = z_real_sq - z_imag_sq + c.real;
        z.imag = 2.0 * z.real * z.imag + c.imag;
        z.real = z_real_temp;
    }

    return iterations; // didn't escape (Max_iterartions - 1)
}

    
